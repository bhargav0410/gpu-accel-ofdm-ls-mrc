#include "hip/hip_runtime.h"

#include "gpuLS.cuh"

/*
	mode:
		= 1 -> master -> creates shared memory 
		= 0 -> slave -> doesn't create the shared memory
*/


//LS
//Y = 16 x 1024
//X = 1 x 1023
//H = 16 x 1023

using namespace std;


gpuLS::gpuLS() {
	//Shared Memory
	std::string shm_uid = shmemID;
	buffPtr = new ShMemSymBuff(shm_uid, mode);
}

gpuLS::~gpuLS() {}

//Reads in Vector X from file -> 1xcols
void gpuLS::matrix_readX(hipFloatComplex* X, int cols){
	ifstream inFile;
	inFile.open(fileNameForX);
	if (!inFile) {
		cerr << "Unable to open file "<< fileNameForX<<", filling in 1+i for x\n";
		float c=1.0f;
		for (int col = 0; col <  cols; col++){
			X[col].x = c;
			X[col].y = c;
		}
		return;
	}
	inFile.read((char*)X, (cols)*sizeof(*X));
	/*
	float c=0;
	for (int col = 0; col <  cols; col++){
		inFile >> c;
		X[col].real=c;
		inFile >> c;
		X[col].imag=c;
	}
	*/
	hipFloatComplex* temp = 0;
	temp=(hipFloatComplex*)malloc ((cols-1)/2* sizeof (*temp));
	//copy second half to temp
	memmove(temp, &X[(cols+1)/2], (cols-1)/2* sizeof (*X));
	//copy first half to second half
	memmove(&X[(cols-1)/2], X, (cols+1)/2* sizeof (*X));
	//copy temp to first half
	memmove(X, temp, (cols-1)/2* sizeof (*X));
	
	free(temp);
	inFile.close();
}

void gpuLS::copyPilotToGPU(hipFloatComplex* dX, int rows, int cols) {
	//X = 1x1023 -> later can become |H|^2
	hipFloatComplex* X = 0;
	int sizeX=rows*(cols-1)* sizeof(*X);
	X = (hipFloatComplex*)malloc(sizeX);
	//hipFloatComplex* H =0;
	//H = (hipFloatComplex *)malloc(sizeX*rows);
	//hipMalloc((void**)&H, size);
	
	//Read in X vector -> 1x1023
	for (int i = 0; i < rows; i++) {
		//std::cout << "Here...\n";
		matrix_readX(&X[i*(cols-1)], cols-1);
	}
	//std::cout << "Here...\n";
	hipMemcpy(dX, X, rows*(cols-1)*sizeof(*dX), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	free(X);
}


__global__ void shiftOneRow(hipFloatComplex* Y, int cols1, int rows1){
	int cols = cols1;
	//int rows = rows1;
	int col = threadIdx.y*cols + threadIdx.x;
	int tid = blockIdx.y*gridDim.x*blockDim.y*cols + blockIdx.x*blockDim.y*cols + threadIdx.y*cols + threadIdx.x;
	extern __shared__ hipFloatComplex temp[];
	
	if ((threadIdx.x + blockIdx.x*blockDim.x) < (cols+1)/2) {
		temp[col] = Y[tid+((cols-1)/2)];
	} else if ((threadIdx.x + blockIdx.x*blockDim.x) >= (cols+1)/2 and (threadIdx.x + blockIdx.x*blockDim.x) < cols) {
		temp[col] = Y[tid-((cols+1)/2)];
	}
	__syncthreads();
	
	Y[tid] = temp[col];
	__syncthreads();
}

void gpuLS::shiftOneRowCPU(hipFloatComplex* Y, int cols, int row){
	hipFloatComplex* Yf = &Y[row*cols];
	//std::cout << "Here...\n";
	hipFloatComplex* temp = 0;
	temp=(hipFloatComplex*)malloc ((cols+1)/2* sizeof (*temp));
	//copy second half to temp
	memmove(temp, &Yf[(cols-1)/2], (cols+1)/2* sizeof (*Yf));
	//copy first half to second half
	memmove(&Yf[(cols+1)/2], Yf, (cols-1)/2* sizeof (*Yf));
	//copy temp to first half
	memmove(Yf, temp, (cols+1)/2* sizeof (*Yf));
	
	free(temp);
	
}

__global__ void dropPrefix(hipFloatComplex *Y, hipFloatComplex *dY, int rows1, int cols1){
	
	int rows = rows1;
	int cols= cols1;
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	Y[tid] = dY[blockIdx.x*(blockDim.x+prefix) + threadIdx.x + prefix];
	/*
	for(int i =0; i<rows; i++){
		memcpy(&Y[i*cols], &dY[i*(cols+prefix)+ prefix], cols*sizeof(*dY));
	}	
	*/
	
	
}

__global__ void findHs(hipFloatComplex* dY, hipFloatComplex* dH, hipFloatComplex* dX, int rows1, int cols1){	
	int cols = cols1-1;
	int rows = rows1;
	int tid = (blockIdx.y*gridDim.x*blockDim.y + blockIdx.x*blockDim.y + threadIdx.y)*cols + threadIdx.x;
	int tid2 = (blockIdx.y*gridDim.x*blockDim.y + blockIdx.x*blockDim.y + threadIdx.y)*(cols+1) + threadIdx.x + 1;
	//find my work
	//Drop first element and copy it into Hconj
	if ((blockIdx.y + threadIdx.y)*blockDim.x + threadIdx.x < cols) {
		dH[tid] = dY[tid2];
	}
	__syncthreads();
	
	//complex division
	//H/X where H = FFT(Y) (w/ dropped first element)
	//Then take conjugate of H
	if (tid < cols*rows) {
		dH[tid] = hipCdivf(dH[tid], dX[tid]);
		dH[tid] = hipConjf(dH[tid]);
		//dX[tid].x = dH[tid].x * dH[tid].x + dH[tid].y * dH[tid].y;
	}
	__syncthreads();
	//Now dH holds conj H
	
	
}


__global__ void findDistSqrd(hipFloatComplex* H, float* Hsqrd, int rows1, int cols1){
	int cols = cols1;
	int rows = rows1;
	//int tid = blockIdx.x*cols + threadIdx.x;
	extern __shared__ hipFloatComplex temp[];
	int sid = threadIdx.x*cols + blockIdx.x*blockDim.y + threadIdx.y;
	int tempID = threadIdx.y*rows + threadIdx.x;
	
	if (sid < rows*cols) {
		temp[tempID] = H[sid];
	}
	temp[tempID].x = temp[tempID].x*temp[tempID].x + temp[tempID].y*temp[tempID].y;
	__syncthreads();
	for (int i = 1; i < rows; i = i*2) {
		if (threadIdx.x%(2*i) == 0 and (blockIdx.x*blockDim.y + threadIdx.y) < cols) {
			temp[tempID].x += temp[tempID+i].x;
		}
		__syncthreads();
	}
	
	
	if(threadIdx.x == 0 and (blockIdx.x*blockDim.y + threadIdx.y) < cols) {
		Hsqrd[blockIdx.x*blockDim.y + threadIdx.y] = temp[tempID].x;
	}
}


__global__ void multiplyWithChannelConj(hipFloatComplex* Y, hipFloatComplex* Hconj, hipFloatComplex* Yf, int rows1, int cols1,int syms1 = 1){
	int rows = rows1;
	int cols = cols1-1;
    int syms = syms1;
    //find my work 
	//Y x conj(H) -> then sum all rows into elements in Hsqrd
	//Y = 16x1024+prefix
	//conjH = 16x1023
	int tid = (blockIdx.z*gridDim.y*gridDim.x*blockDim.y + blockIdx.x*blockDim.y + threadIdx.y)*cols + blockIdx.y*blockDim.x + threadIdx.x;
	int tid2 = (blockIdx.z*gridDim.y*gridDim.x*blockDim.y + blockIdx.x*blockDim.y + threadIdx.y)*(cols+1) + blockIdx.y*blockDim.x + threadIdx.x + 1;
	int hid = blockIdx.x*blockDim.y*cols + blockIdx.y*blockDim.x + threadIdx.y*cols + threadIdx.x;
	
	if (blockIdx.y*blockDim.x + threadIdx.x < cols) {
		Yf[tid] = Y[tid2];
	}
	__syncthreads();
	
	if (tid < rows*cols*syms) {
		Yf[tid] = hipCmulf(Yf[tid],Hconj[hid]);
	}
	__syncthreads();
}


__global__ void combineForMRC(hipFloatComplex* Y, float* Hsqrd, int rows1, int cols1) {
	int rows = rows1;
	int cols = cols1;
	int row = blockIdx.x*blockDim.y + threadIdx.y;
	int col = threadIdx.x;
	//int tid = blockIdx.x*blockDim.x + threadIdx.x;
	extern __shared__ hipFloatComplex temp[];
	int tempID = threadIdx.y*rows + threadIdx.x;
	int sid = blockIdx.y*rows*cols + threadIdx.x*cols + blockIdx.x*blockDim.y + threadIdx.y;
	temp[tempID] = Y[sid];
	
	for (int i = 1; i < rows; i = i*2) {
		if (threadIdx.x%(2*i) == 0 and row < cols) {
			temp[tempID] = hipCaddf(temp[tempID],temp[tempID+i]);
		}
		__syncthreads();
	}
	
	if (threadIdx.x == 0 and row < cols) {
		Y[row + cols*blockIdx.y].x = temp[tempID].x/Hsqrd[row];
		Y[row + cols*blockIdx.y].y = temp[tempID].y/Hsqrd[row];
		__syncthreads();
	}
}

/*-----------------------------------GPU kernel calling functions--------------------------------------*/

void gpuLS::ShiftOneRow(hipFloatComplex* Y, int cols1, int rows1, dim3 blockDim, dim3 gridDim, hipStream_t* stream) {
	hipStream_t localStreamVar = *stream; 
	shiftOneRow<< <gridDim, blockDim, 0, localStreamVar>> >(Y, cols1, rows1);
}

void gpuLS::DropPrefix(hipFloatComplex *Y, hipFloatComplex *dY, int rows1, int cols1, dim3 blockDim, dim3 gridDim, hipStream_t* stream) {
	hipStream_t localStreamVar = *stream;
	dropPrefix<< <gridDim, blockDim, 0, localStreamVar>> >(Y, dY, rows1, cols1);
}

void gpuLS::FindLeastSquaresGPU(hipFloatComplex* dY, hipFloatComplex* dH, hipFloatComplex* dX, int rows1, int cols1, dim3 blockDim, dim3 gridDim, hipStream_t* stream) {
	hipStream_t localStreamVar = *stream;
	findHs<< <gridDim, blockDim, 0, localStreamVar>> >(dY, dH, dX, rows1, cols1);
}

void gpuLS::FindHsqrdforMRC(hipFloatComplex* H, float* Hsqrd, int rows1, int cols1, dim3 blockDim, dim3 gridDim, hipStream_t* stream) {
	hipStream_t localStreamVar = *stream;
	size_t sharedMemSize = blockDim.x*blockDim.y*blockDim.z;
	findDistSqrd<< <gridDim, blockDim, sharedMemSize, localStreamVar>> >(H, Hsqrd, rows1, cols1);
}

void gpuLS::MultiplyWithChannelConj(hipFloatComplex* Y, hipFloatComplex* Hconj, hipFloatComplex* Yf, int rows1, int cols1, int syms1, dim3 blockDim, dim3 gridDim, hipStream_t* stream) {
	hipStream_t localStreamVar = *stream;
	multiplyWithChannelConj<< <gridDim, blockDim, 0, localStreamVar>> >(Y, Hconj, Yf, rows1, cols1, syms1);
}

void gpuLS::CombineForMRC(hipFloatComplex* Y, float* Hsqrd, int rows1, int cols1, dim3 blockDim, dim3 gridDim, hipStream_t* stream) {
	hipStream_t localStreamVar = *stream;
	size_t sharedMemSize = blockDim.x*blockDim.y*blockDim.z;
	combineForMRC<< <gridDim, blockDim, sharedMemSize, localStreamVar>> >(Y, Hsqrd, rows1, cols1);
}

/*-----------------------------------CuBlas based functions--------------------------------------*/

__global__ void findDistSqrdCuBlas(hipFloatComplex* H, float* Hsqrd, int rows1, int cols1) {
	int rows = rows1;
	int cols = cols1;
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	
	if (tid < cols) {
		hipblasScnrm2(handle, rows, &H[tid], cols, &Hsqrd[tid]);
	}
	
	hipblasDestroy(handle);
}

__global__ void multiplyWithChanEstCuBlas(hipFloatComplex* Y, hipFloatComplex* Hconj, hipFloatComplex* Yf, float* Hsqrd, int rows1, int cols1, int syms1 = 1) {
	int rows = rows1, cols = cols1-1, syms = syms1;
	int tid = blockIdx.z*gridDim.y*blockDim.y*cols + blockIdx.y*blockDim.x + threadIdx.y*cols + threadIdx.x;
	int tid2 = blockIdx.z*gridDim.y*blockDim.y*(cols+1)*rows + blockIdx.y*blockDim.x + threadIdx.y*(cols+1) + threadIdx.x + 1;
	int hid = blockIdx.y*blockDim.x + threadIdx.y*cols + threadIdx.x;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	
	/*
	if ((blockIdx.y + threadIdx.y)*cols + threadIdx.x < cols) {
		Yf[tid] = Y[tid2];
	}
	__syncthreads();
	*/
	
	if (tid2 < rows*(cols+1)*syms) {
		hipblasCdotc(handle, rows, &Hconj[hid], cols,  &Y[tid2], cols + 1, &Yf[tid]);;
	}
	__syncthreads();
	
	if (tid < cols*syms) {
		Yf[tid].x = Yf[tid].x/Hsqrd[hid];
		Yf[tid].y = Yf[tid].y/Hsqrd[hid];
	}
	
	hipblasDestroy(handle);
	
}

/*-----------------------------------Host Functions--------------------------------------*/

void gpuLS::batchedFFT(hipFloatComplex* Y, int rows, int cols, hipStream_t* stream) {
	hipStream_t localStreamVar = *stream;
	hipfftHandle plan;
	hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	cufftSeetStream(plan, localStreamVar);
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
}

void gpuLS::firstVector(hipFloatComplex* dY, hipFloatComplex* Y, hipFloatComplex* dH, hipFloatComplex* dX, float* Hsqrd, int rows, int cols, int it){
	clock_t start, finish;
	//std::cout << "Here...\n";	
	
	// CUFFT plan -> do it one time before?
	
	
	//Read in Y with prefix
	buffPtr->readNextSymbolCUDA(dY, it);
	
	if(timerEn){
		start = clock();
	}
	hipMemcpy(Y, dY, rows*cols*sizeof(*Y), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	if(timerEn){
		finish = clock();
		buffPtr->setReadT(((float)(finish - start))/(float)CLOCKS_PER_SEC, it);
	}
	
	if(timerEn){
		start = clock();
	}
	
	
	//FFT(Y)
	hipfftHandle plan;
	hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	hipfftDestroy(plan);
	hipDeviceSynchronize();
	if(timerEn){
		finish = clock();
		buffPtr->setFft(((float)(finish - start))/(float)CLOCKS_PER_SEC, it);
	}
	
	//find Hconj and Hsqrd
	if(timerEn){
		start = clock();
	}
//	dim3 dimBlock(numOfBlocks, threadsPerBlock-1);
	findHs<< <numOfBlocks, threadsPerBlock-1>> >(Y, dH, dX, rows, cols);
	hipDeviceSynchronize();
	//Save |H|^2 into Hsqrd
	findDistSqrd<< <threadsPerBlock-1, numOfBlocks, numOfBlocks*sizeof(hipFloatComplex)>> >(dH, Hsqrd, rows, cols-1);
	hipDeviceSynchronize();
	
	if(timerEn){
		finish = clock();
		buffPtr->setDecode(((float)(finish - start))/(float)CLOCKS_PER_SEC, it);
	}
	
	//free(X);
	//hipFree(H);
	
	//dH holds H conj
	//dX holds {H^2)	
}

void gpuLS::demodOneSymbol(hipFloatComplex *dY, hipFloatComplex* Y, hipFloatComplex *Hconj, float *Hsqrd,int rows1, int cols1, int it) {
	int rows = rows1;
	int cols= cols1;

	clock_t start, finish;
	//Y x conj(H) -> then sum all rows into elements in Hsqrd
	//Y = 16x1024+prefix
	//conjH = 16x1024
	
	if(it==numberOfSymbolsToTest-1){
		//if last one
		buffPtr->readLastSymbolCUDA(dY);
	} else {
		buffPtr->readNextSymbolCUDA(dY, it);
	}
	
	if(timerEn){
		start = clock();
	}
	hipMemcpy(Y, dY, rows*cols*sizeof(*Y), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	if(timerEn){
		finish = clock();
		buffPtr->setReadT(((float)(finish - start))/(float)CLOCKS_PER_SEC, it);
	}
	
	if(timerEn){
		start = clock();
	}
	
	//FFT(Y)
	hipfftHandle plan;
	hipfftPlan1d(&plan, threadsPerBlock, HIPFFT_C2C, numOfBlocks);
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	hipfftDestroy(plan);
	hipDeviceSynchronize();
	if(timerEn){
		finish = clock();
		buffPtr->setFft(((float)(finish - start))/(float)CLOCKS_PER_SEC, it);
	}
	
	hipFloatComplex* Yf = 0;
	hipMalloc((void**)&Yf, rows*(cols-1)* sizeof (*Yf));
	
	if(timerEn){
		start = clock();
	}
	multiplyWithChannelConj<< <numOfBlocks, threadsPerBlock-1>> >(Y, Hconj, Yf, rows, cols);
	hipDeviceSynchronize();
	combineForMRC<< <threadsPerBlock-1, numOfBlocks, numOfBlocks*sizeof(hipFloatComplex)>> >(Yf, Hsqrd, rows, cols-1);
	hipDeviceSynchronize();
	hipMemcpy(dY, Yf, (cols-1)*sizeof(*dY), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	
	shiftOneRowCPU(dY,cols-1,0);
	
	if(timerEn){
		finish = clock();
		buffPtr->setDecode(((float)(finish - start))/(float)CLOCKS_PER_SEC, it);
	}
	
	hipFree(Yf);
	hipDeviceSynchronize();
}

void gpuLS::demodOneFrame(hipFloatComplex *dY, hipFloatComplex* Y, hipFloatComplex* dX, hipFloatComplex *Hconj, float *Hsqrd, int rows1, int cols1) {
	int rows = rows1;
	int cols = cols1;
	int maxThreads = devProp.maxThreadsPerBlock;
	clock_t start, finish;
	//Y x conj(H) -> then sum all rows into elements in Hsqrd
	//Y = 16x1024+prefix
	//conjH = 16x1024
	
	for (int it = 0; it < numberOfSymbolsToTest; it++) {
		if(it==numberOfSymbolsToTest-1){
			//if last one
			buffPtr->readLastSymbol(&dY[rows*cols*it]);
		} else {
			buffPtr->readNextSymbol(&dY[rows*cols*it], it);
		}
	}
	
	if(timerEn){
		start = clock();
	}
	hipMemcpy(Y, dY, rows*cols*(lenOfBuffer)*sizeof(*Y), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	if(timerEn){
		finish = clock();
		buffPtr->setReadT(((float)(finish - start))/(float)CLOCKS_PER_SEC, 1);
	}
	
	if(timerEn){
		start = clock();
	}
	
	//FFT(Y)
	hipfftHandle plan;
	hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows*(lenOfBuffer));
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	hipfftDestroy(plan);
	hipDeviceSynchronize();
	if(timerEn){
		finish = clock();
		buffPtr->setFft(((float)(finish - start))/(float)CLOCKS_PER_SEC, 1);
	}
	
	
	if(timerEn){
		start = clock();
	}
//	dim3 dimBlock(numOfBlocks, threadsPerBlock-1);
	if (threadsPerBlock <= maxThreads) {
		findHs<< <numOfBlocks, threadsPerBlock-1>> >(Y, Hconj, dX, rows, cols);
		hipDeviceSynchronize();
		//Save |H|^2 into Hsqrd
	} else {
		dim3 chanEstDim(numOfBlocks,ceil(threadsPerBlock/maxThreads));
		findHs<< <chanEstDim, maxThreads>> >(Y, Hconj, dX, rows, cols);
		hipDeviceSynchronize();
		//Save |H|^2 into Hsqrd
	}
	findDistSqrd<< <threadsPerBlock-1, numOfBlocks, numOfBlocks*sizeof(hipFloatComplex)>> >(Hconj, Hsqrd, rows, cols-1);
	hipDeviceSynchronize();
	
	if(timerEn){
		finish = clock();
		buffPtr->setDecode(((float)(finish - start))/(float)CLOCKS_PER_SEC, 0);
	}
	
	if(timerEn){
		start = clock();
	}
	hipFloatComplex* Yf = 0;
	hipMalloc((void**)&Yf, rows*(cols-1)*(lenOfBuffer-1)* sizeof (*Yf));
	if (threadsPerBlock <= maxThreads) {
		dim3 gridDims1(numOfBlocks, 1, lenOfBuffer-1);
		multiplyWithChannelConj<< <gridDims1, threadsPerBlock-1>> >(&Y[rows*cols], Hconj, Yf, rows, cols, numberOfSymbolsToTest-1);
		hipDeviceSynchronize();
	} else {
		dim3 gridDims1(numOfBlocks, ceil(threadsPerBlock/maxThreads), lenOfBuffer-1);
		multiplyWithChannelConj<< <gridDims1, maxThreads>> >(&Y[rows*cols], Hconj, Yf, rows, cols, numberOfSymbolsToTest-1);
		hipDeviceSynchronize();
	}
	dim3 gridDims2(threadsPerBlock-1, lenOfBuffer-1);
	combineForMRC<< <gridDims2, numOfBlocks, numOfBlocks*sizeof(hipFloatComplex)>> >(Yf, Hsqrd, rows, cols-1);
	hipDeviceSynchronize();
	shiftOneRow<< <lenOfBuffer-1, threadsPerBlock-1, (threadsPerBlock-1)*sizeof(hipFloatComplex)>> >(Yf, cols-1, rows);
	hipDeviceSynchronize();
	hipMemcpy(dY, Yf, (cols-1)*(lenOfBuffer-1)*sizeof(*dY), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	/*
	for (int it = 1; it < lenOfBuffer-1; it++) {
		shiftOneRowCPU(dY,cols-1,0);
	}
	*/
	if(timerEn){
		finish = clock();
		buffPtr->setDecode(((float)(finish - start))/(float)CLOCKS_PER_SEC, 1);
	}
	hipFree(Yf);
	hipDeviceSynchronize();
}

void gpuLS::demodOneFrameCUDA(hipFloatComplex* dY, hipFloatComplex* Y, hipFloatComplex* dX, hipFloatComplex *Hconj, float *Hsqrd, int rows1, int cols1) {
	int rows = rows1;
	int cols = cols1;
	int maxThreads = devProp.maxThreadsPerBlock;
	
	clock_t start, finish;
	//Y x conj(H) -> then sum all rows into elements in Hsqrd
	//Y = 16x1024+prefix
	//conjH = 16x1024
	/*
	for (int it = 0; it < numberOfSymbolsToTest; it++) {
		if(it==numberOfSymbolsToTest-1){
			//if last one
			buffPtr->readLastSymbolCUDA(&Y[rows*cols*it]);
		} else {
			buffPtr->readNextSymbolCUDA(&Y[rows*cols*it], it);
		}
	}
//	hipDeviceSynchronize();
	*/
	if(timerEn){
		start = clock();
	}
	
	//FFT(Y)
	hipfftHandle plan;
	hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows*(lenOfBuffer));
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	hipfftDestroy(plan);
	hipDeviceSynchronize();
	if(timerEn){
		finish = clock();
		buffPtr->setFft(((float)(finish - start))/(float)CLOCKS_PER_SEC, 1);
	}
	
	
	if(timerEn){
		start = clock();
	}
//	dim3 dimBlock(numOfBlocks, threadsPerBlock-1);

	if (threadsPerBlock <= maxThreads) {
		findHs<< <numOfBlocks, threadsPerBlock>> >(Y, Hconj, dX, rows, cols);
		hipDeviceSynchronize();
		//Save |H|^2 into Hsqrd
	} else {
		dim3 chanEstBlockDim1(maxThreads);
		dim3 chanEstGridDim1(numOfBlocks,ceil((float)threadsPerBlock/(float)maxThreads));
		findHs<< <chanEstGridDim1, chanEstBlockDim1>> >(Y, Hconj, dX, rows, cols);
	//	hipDeviceSynchronize();
		//Save |H|^2 into Hsqrd
	}
//	dim3 chanEstBlockDim2(rows,ceil((float)maxThreads/(float)rows));
//	dim3 chanEstGridDim2(ceil((float)(cols)/(ceil((float)maxThreads/(float)rows))),ceil((float)rows/(float)maxThreads));
	findDistSqrd<< <threadsPerBlock-1,numOfBlocks, numOfBlocks*sizeof(hipFloatComplex)>> >(Hconj, Hsqrd, rows, cols-1);
	hipDeviceSynchronize();
	
	if(timerEn){
		finish = clock();
		buffPtr->setDecode(((float)(finish - start))/(float)CLOCKS_PER_SEC, 0);
	}
	
	if(timerEn){
		start = clock();
	}
	hipFloatComplex* Yf = 0;
	hipMalloc((void**)&Yf, rows*(cols-1)*(lenOfBuffer-1)* sizeof (*Yf));
	if (threadsPerBlock <= maxThreads) {
		dim3 gridDims1(numOfBlocks, 1, lenOfBuffer-1);
		multiplyWithChannelConj<< <gridDims1, threadsPerBlock-1>> >(&Y[rows*cols], Hconj, Yf, rows, cols, numberOfSymbolsToTest-1);
		hipDeviceSynchronize();
	} else {
		dim3 gridDims1(numOfBlocks, ceil((float)threadsPerBlock/(float)maxThreads), lenOfBuffer-1);
		multiplyWithChannelConj<< <gridDims1, maxThreads>> >(&Y[rows*cols], Hconj, Yf, rows, cols, numberOfSymbolsToTest-1);
		hipDeviceSynchronize();
	}
	dim3 gridDims2(threadsPerBlock-1, lenOfBuffer-1);
	combineForMRC<< <gridDims2, numOfBlocks, numOfBlocks*sizeof(hipFloatComplex)>> >(Yf, Hsqrd, rows, cols-1);
	hipDeviceSynchronize();
	if (threadsPerBlock <= maxThreads) {
		dim3 gridDims3(1,lenOfBuffer-1);
		shiftOneRow<< <gridDims3, threadsPerBlock-1, (threadsPerBlock-1)*sizeof(hipFloatComplex)>> >(Yf, cols-1, rows);
		hipDeviceSynchronize();
	} else {
		dim3 gridDims3(ceil((float)threadsPerBlock/(float)maxThreads), lenOfBuffer-1);
		shiftOneRow<< <gridDims3, maxThreads, (threadsPerBlock-1)*sizeof(hipFloatComplex)>> >(Yf, cols-1, rows);
		hipDeviceSynchronize();
	}
	hipMemcpy(dY, Yf, (cols-1)*(lenOfBuffer-1)*sizeof(*dY), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	/*
	for (int it = 1; it < lenOfBuffer-1; it++) {
		shiftOneRowCPU(dY,cols-1,0);
	}
	*/
	if(timerEn){
		finish = clock();
		buffPtr->setDecode(((float)(finish - start))/(float)CLOCKS_PER_SEC, 1);
	}
	hipFree(Yf);
}

void gpuLS::demodOptimized(hipFloatComplex* dY, hipFloatComplex* Y, hipFloatComplex* dX, hipFloatComplex *Hconj, float *Hsqrd, int rows1, int cols1) {
//	hipblasHandle_t handle;
//	hipblasCreate(&handle);
	
	int rows = rows1;
	int cols = cols1;
	int maxThreads = devProp.maxThreadsPerBlock;
	clock_t start, finish;
	//Y x conj(H) -> then sum all rows into elements in Hsqrd
	//Y = 16x1024+prefix
	//conjH = 16x1024
//	hipDeviceSynchronize();
	
	if(timerEn){
		start = clock();
	}
	
	//FFT(Y)
	hipfftHandle plan;
	hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows*(lenOfBuffer));
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	hipfftDestroy(plan);
//	hipDeviceSynchronize();
	if(timerEn){
		finish = clock();
		buffPtr->setFft(((float)(finish - start))/(float)CLOCKS_PER_SEC, 1);
	}
	
	
	if(timerEn){
		start = clock();
	}
	
	if (threadsPerBlock <= maxThreads) {
		dim3 chanEstBlockDim(cols,ceil((float)maxThreads/(float)cols));
		dim3 chanEstGridDim(ceil((float)rows/ceil((float)maxThreads/(float)cols)),1);
		findHs<< <chanEstGridDim, chanEstBlockDim>> >(Y, Hconj, dX, rows, cols);
		hipDeviceSynchronize();
		//Save |H|^2 into Hsqrd
	} else {
		dim3 chanEstBlockDim1(maxThreads);
		dim3 chanEstGridDim1(rows,ceil((float)cols/(float)maxThreads));
		findHs<< <chanEstGridDim1, chanEstBlockDim1>> >(Y, Hconj, dX, rows, cols);
		hipDeviceSynchronize();
		//Save |H|^2 into Hsqrd
	}
	
	dim3 distSqrdBlockDim(rows,ceil((float)maxThreads/(float)rows));
	dim3 distSqrdGridDim(ceil((float)(cols-1)/ceil((float)maxThreads/(float)rows)),1);
	findDistSqrd<< <distSqrdGridDim, distSqrdBlockDim, maxThreads*sizeof(hipFloatComplex)>> >(Hconj, Hsqrd, rows, cols-1);
	
	if(timerEn){
		finish = clock();
		buffPtr->setDecode(((float)(finish - start))/(float)CLOCKS_PER_SEC, 0);
	}
	
	if(timerEn){
		start = clock();
	}
	hipFloatComplex* Yf = 0;
	hipMalloc((void**)&Yf, rows*(cols-1)*(lenOfBuffer-1)* sizeof (*Yf));
	if (threadsPerBlock <= maxThreads) {
		dim3 blockDims1(cols,ceil((float)maxThreads/(float)cols));
		dim3 gridDims1(ceil((float)rows/ceil((float)maxThreads/(float)cols)), 1, lenOfBuffer-1);
		multiplyWithChannelConj<< <gridDims1, blockDims1>> >(&Y[rows*cols], Hconj, Yf, rows, cols, numberOfSymbolsToTest-1);
		hipDeviceSynchronize();
	} else {
		dim3 gridDims1(rows, ceil((float)cols/(float)maxThreads), lenOfBuffer-1);
		multiplyWithChannelConj<< <gridDims1, maxThreads>> >(&Y[rows*cols], Hconj, Yf, rows, cols, numberOfSymbolsToTest-1);
		hipDeviceSynchronize();
	}
	
	dim3 blockDims2(rows,ceil((float)maxThreads/(float)rows));
	dim3 gridDims2(ceil((float)(cols-1)/ceil((float)maxThreads/(float)rows)), lenOfBuffer-1);
	combineForMRC<< <gridDims2, blockDims2, maxThreads*sizeof(hipFloatComplex)>> >(Yf, Hsqrd, rows, cols-1);
	hipDeviceSynchronize();
	if (cols <= maxThreads) {
		dim3 gridDims3(1,lenOfBuffer-1);
		shiftOneRow<< <gridDims3, cols-1, (cols-1)*sizeof(hipFloatComplex)>> >(Yf, cols-1, rows);
		hipDeviceSynchronize();
	} else {
		dim3 gridDims3(ceil((float)cols/(float)maxThreads), lenOfBuffer-1);
		shiftOneRow<< <gridDims3, maxThreads, (cols-1)*sizeof(hipFloatComplex)>> >(Yf, cols-1, rows);
		hipDeviceSynchronize();
	}
	hipMemcpy(dY, Yf, (cols-1)*(lenOfBuffer-1)*sizeof(*dY), hipMemcpyDeviceToHost);
	
	if(timerEn){
		finish = clock();
		buffPtr->setDecode(((float)(finish - start))/(float)CLOCKS_PER_SEC, 1);
	}
	hipFree(Yf);
}

void gpuLS::demodCuBlas(hipFloatComplex* dY, hipFloatComplex* Y, hipFloatComplex* dX, hipFloatComplex *Hconj, float *Hsqrd, int rows1, int cols1) {
//	hipblasHandle_t handle;
//	hipblasCreate(&handle);
	
	int rows = rows1;
	int cols = cols1;
	int maxThreads = devProp.maxThreadsPerBlock;
	clock_t start, finish;
	//Y x conj(H) -> then sum all rows into elements in Hsqrd
	//Y = 16x1024+prefix
	//conjH = 16x1024
//	hipDeviceSynchronize();
	
	if(timerEn){
		start = clock();
	}
	
	//FFT(Y)
	hipfftHandle plan;
	hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows*(lenOfBuffer));
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	hipfftDestroy(plan);
//	hipDeviceSynchronize();
	if(timerEn){
		finish = clock();
		buffPtr->setFft(((float)(finish - start))/(float)CLOCKS_PER_SEC, 1);
	}
	
	
	if(timerEn){
		start = clock();
	}
	
	if (cols <= maxThreads) {
		dim3 chanEstBlockDim(cols,ceil((float)maxThreads/(float)cols));
		dim3 chanEstGridDim(ceil((float)rows/ceil((float)maxThreads/(float)cols)),1);
		findHs<< <chanEstGridDim, chanEstBlockDim>> >(Y, Hconj, dX, rows, cols);
		findDistSqrdCuBlas<< <1, cols-1>> >(Hconj, Hsqrd, rows, cols-1);
	//	hipDeviceSynchronize();
		//Save |H|^2 into Hsqrd
	} else {
		dim3 chanEstBlockDim1(maxThreads);
		dim3 chanEstGridDim1(rows,ceil((float)cols/(float)maxThreads));
		findHs<< <chanEstGridDim1, chanEstBlockDim1>> >(Y, Hconj, dX, rows, cols);
		findDistSqrdCuBlas<< <ceil((float)cols/(float)maxThreads), chanEstBlockDim1>> >(Hconj, Hsqrd, rows, cols-1);

	//	hipDeviceSynchronize();
		//Save |H|^2 into Hsqrd
	}
	
		
	if(timerEn){
		finish = clock();
		buffPtr->setDecode(((float)(finish - start))/(float)CLOCKS_PER_SEC, 0);
	}
	
	if(timerEn){
		start = clock();
	}
	hipFloatComplex* Yf = 0;
	hipMalloc((void**)&Yf, rows*(cols-1)*(lenOfBuffer-1)* sizeof (*Yf));
	if (cols <= maxThreads) {
		dim3 blockDims1(cols,ceil((float)maxThreads/(float)cols));
		dim3 gridDims1(1, 1, ceil((float)lenOfBuffer-1/(ceil((float)maxThreads/(float)cols))));
		multiplyWithChanEstCuBlas<< <gridDims1, blockDims1>> >(&Y[rows*cols], Hconj, Yf, Hsqrd, rows, cols, numberOfSymbolsToTest-1);
	//	hipDeviceSynchronize();
	} else {
		dim3 gridDims1(1, ceil((float)cols/(float)maxThreads), lenOfBuffer-1);
		multiplyWithChanEstCuBlas<< <gridDims1, maxThreads>> >(&Y[rows*cols], Hconj, Yf, Hsqrd, rows, cols, numberOfSymbolsToTest-1);
	//	hipDeviceSynchronize();
	}
	if (cols <= maxThreads) {
		dim3 gridDims3(1,lenOfBuffer-1);
		shiftOneRow<< <gridDims3, cols-1, (cols-1)*sizeof(hipFloatComplex)>> >(Yf, cols-1, rows);
		hipDeviceSynchronize();
	} else {
		dim3 gridDims3(ceil((float)cols/(float)maxThreads), lenOfBuffer-1);
		shiftOneRow<< <gridDims3, maxThreads, (cols-1)*sizeof(hipFloatComplex)>> >(Yf, cols-1, rows);
		hipDeviceSynchronize();
	}
	hipMemcpy(dY, Yf, (cols-1)*(lenOfBuffer-1)*sizeof(*dY), hipMemcpyDeviceToHost);
	
	if(timerEn){
		finish = clock();
		buffPtr->setDecode(((float)(finish - start))/(float)CLOCKS_PER_SEC, 1);
	}
	hipFree(Yf);
}