#include "hip/hip_runtime.h"
#ifndef cudaEn
	#define cudaEn
#endif

//Shared Memory 
#include "ShMemSymBuff_cucomplex.hpp"
#include "gpuLS.cuh"
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>
#define FFT_size dimension
#define cp_size prefix
#define numSymbols lenOfBuffer


/*
	mode:
		= 1 -> master -> creates shared memory 
		= 0 -> slave -> doesn't create the shared memory
*/
 
//!How to Compile:   nvcc ../../examples/gpuLS_cucomplex.cu -lcufft -lrt -o gpu -arch=sm_35
// ./gpu

//LS
//Y = 16 x 1024
//X = 1 x 1023
//H = 16 x 1023

using namespace std;

std::string file = "Output_gpu.dat";
//std::ofstream outfile;

int main(){
	int rows = numOfRows; // number of vectors
	int cols=dimension;//dimension
	hipSetDevice(0);
	//printf("CUDA LS: \n");
	//printInfo();
	//dY holds symbol with prefix
	hipFloatComplex *dY = 0;
	dY = (hipFloatComplex*)malloc(rows*(cols)* sizeof (*dY));
	
	float *Hsqrd = 0;
	hipMalloc((void**)&Hsqrd, (cols-1)* sizeof (*Hsqrd));
	
	//dH (and Hconj) = 16x1023
	hipFloatComplex *dH = 0;
	hipMalloc((void**)&dH, rows*(cols-1)* sizeof (*dH));
	
	//X = 1x1023 -> later can become |H|^2
	hipFloatComplex *dX = 0;
	hipMalloc((void**)&dX, rows*(cols-1)* sizeof (*dX));
	
	hipFloatComplex *Yf = 0;
	Yf = (hipFloatComplex*)malloc((cols-1)* sizeof (*Yf));
	
	hipFloatComplex* Y = 0;
	hipMalloc((void**)&Y, rows*cols*sizeof(*Y));
	
	
	hipfftHandle plan;
	hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	
	
	//Shared Memory
	//string shm_uid = shmemID;
	//gpu->buffPtr = new ShMemSymBuff(shm_uid, mode);
	
	copyPilotToGPU(dX, rows, cols);
	
	for (int iter = 0; iter < numTimes; iter++) {
	firstVector(dY, Y, dH, dX, Hsqrd, rows, cols, 0);
	//dH holds h conj
	//dX holds |H|^2
	for(int i=1; i<numberOfSymbolsToTest; i++){
		
		demodOneSymbol(dY, Y, dH, Hsqrd, rows, cols, i);
		
		if(testEn){
			//printf("Symbol #%d:\n", i);
			//cuda copy it over
			memcpy(Yf, dY, (cols-1)* sizeof (*Yf));
			if (i <= 1) {
				outfile.open(file.c_str(), std::ofstream::binary | std::ofstream::trunc);
			} else {
				outfile.open(file.c_str(), std::ofstream::binary | std::ofstream::app);
			}
			outfile.write((const char*)Yf, (cols-1)*sizeof(*Yf));
			outfile.close();
			//printOutArr(Yf, 1, cols-1);
		}
		
		
	}
	}
	free(Yf);
	free(dY);
	hipFree(Y);
	hipFree(dH);
	hipFree(dX);
	hipFree(Hsqrd);
	//delete buffPtr;
	
	if(timerEn) {
		printTimes(true);
		storeTimes(false);
	}
	return 0;

}