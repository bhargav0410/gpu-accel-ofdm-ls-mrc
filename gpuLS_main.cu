#include "hip/hip_runtime.h"
/*
Copyright (c) 2018, WINLAB, Rutgers University, USA
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are met:

* Redistributions of source code must retain the above copyright notice, this
  list of conditions and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright notice,
  this list of conditions and the following disclaimer in the documentation
  and/or other materials provided with the distribution.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#ifndef cudaEn
	#define cudaEn
#endif

//Shared Memory 
#include "ShMemSymBuff_cucomplex.hpp"
#include "gpuLS.cuh"
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <assert.h>
#define FFT_size dimension
#define cp_size prefix
#define numSymbols lenOfBuffer


/*
	mode:
		= 1 -> master -> creates shared memory 
		= 0 -> slave -> doesn't create the shared memory
*/
 
//!How to Compile:   nvcc ../../examples/gpuLS_cucomplex.cu -lcufft -lrt -o gpu -arch=sm_35
// ./gpu

//LS
//Y = 16 x 1024
//X = 1 x 1023
//H = 16 x 1023

using namespace std;

std::string file = "Output_gpu.dat";
//std::ofstream outfile;

int main(){
	int rows = numOfRows; // number of vectors
	int cols=dimension;//dimension
	hipSetDevice(0);
	//printf("CUDA LS: \n");
	//printInfo();
	//dY holds symbol with prefix
	hipFloatComplex *dY = 0;
	dY = (hipFloatComplex*)malloc(rows*(cols)* sizeof (*dY));
	
	float *Hsqrd = 0;
	hipMalloc((void**)&Hsqrd, (cols-1)* sizeof (*Hsqrd));
	
	//dH (and Hconj) = 16x1023
	hipFloatComplex *dH = 0;
	hipMalloc((void**)&dH, rows*(cols-1)* sizeof (*dH));
	
	//X = 1x1023 -> later can become |H|^2
	hipFloatComplex *dX = 0;
	hipMalloc((void**)&dX, rows*(cols-1)* sizeof (*dX));
	
	hipFloatComplex *Yf = 0;
	Yf = (hipFloatComplex*)malloc((cols-1)* sizeof (*Yf));
	
	hipFloatComplex* Y = 0;
	hipMalloc((void**)&Y, rows*cols*sizeof(*Y));
	
	
	hipfftHandle plan;
	hipfftPlan1d(&plan, cols, HIPFFT_C2C, rows);
	hipfftExecC2C(plan, (hipfftComplex *)Y, (hipfftComplex *)Y, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	
	
	//Shared Memory
	//string shm_uid = shmemID;
	//gpu->buffPtr = new ShMemSymBuff(shm_uid, mode);
	
	copyPilotToGPU(dX, rows, cols);
	
	for (int iter = 0; iter < numTimes; iter++) {
	firstVector(dY, Y, dH, dX, Hsqrd, rows, cols, 0);
	//dH holds h conj
	//dX holds |H|^2
	for(int i=1; i<numberOfSymbolsToTest; i++){
		
		demodOneSymbol(dY, Y, dH, Hsqrd, rows, cols, i);
		
		if(testEn){
			//printf("Symbol #%d:\n", i);
			//cuda copy it over
			memcpy(Yf, dY, (cols-1)* sizeof (*Yf));
			if (i <= 1) {
				outfile.open(file.c_str(), std::ofstream::binary | std::ofstream::trunc);
			} else {
				outfile.open(file.c_str(), std::ofstream::binary | std::ofstream::app);
			}
			outfile.write((const char*)Yf, (cols-1)*sizeof(*Yf));
			outfile.close();
			//printOutArr(Yf, 1, cols-1);
		}
		
		
	}
	}
	free(Yf);
	free(dY);
	hipFree(Y);
	hipFree(dH);
	hipFree(dX);
	hipFree(Hsqrd);
	//delete buffPtr;
	
	if(timerEn) {
		printTimes(true);
		storeTimes(false);
	}
	return 0;

}